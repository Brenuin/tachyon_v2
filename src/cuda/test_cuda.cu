#include <iostream>
#include <hip/hip_runtime.h>

__global__ void cudaHelloKernel() {
    printf("Hello from the CUDA kernel! (thread %d)\n", threadIdx.x);
}

void launchCudaHello() {
    std::cout << "Launching CUDA kernel... with glfw" << std::endl;

    // Launch with 1 block of 4 threads for demonstration
    cudaHelloKernel<<<1, 4>>>();
    
    // Ensure output is flushed 
    hipDeviceSynchronize();
}
